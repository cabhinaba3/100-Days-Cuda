
#include <hip/hip_runtime.h>
#include<iostream>
// In this example we will implement a simple Matrix x Vector

__global__ void vectorMatrixMultiplication(const float *A, const float *B, float* C,int size){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < size){
        float sum = 0.0f;
        for(int j=0;j<size;j++){ sum += A[index * size +j] * B[j]; }
        C[index] = sum;
    }
}
int main(){
    const int size = 5;
    int matrixSize = size * size * sizeof(float);
    int vectorSize = size * sizeof(float);
    float *A, *B, *C;

    // Initialize the input metrics
    A = (float *)malloc(matrixSize);
    B = (float *)malloc(vectorSize);
    C = (float *)malloc(vectorSize);

    for(int i=0;i<size; ++i){
        for(int j=0;j<size; ++j){
            A[i * size + j] = (i+j) % 100;
        }
        B[i] = i % 50;
        C[i] = 0.0;
    }

    float *d_a, *d_b, *d_c;
    // allocate memory for matrix, vector and output vector
    hipMalloc(&d_a, matrixSize);
    hipMalloc(&d_b, vectorSize);
    hipMalloc(&d_c, vectorSize);

    // copy the Matrix A and matrix B to GPU
    hipMemcpy(d_a, A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, vectorSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ( size + blockSize - 1) / blockSize;
    vectorMatrixMultiplication<<<gridSize, blockSize>>>(d_a, d_b, d_c, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        // Likely PTX for my GPU(check)
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    // synchronize the device and host
    hipDeviceSynchronize();
    hipMemcpy(C, d_c, vectorSize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) { printf("%.2f ", A[i * size + j]); }
        printf("\n");
    }
    for (int i = 0; i < size; i++) { printf("%f ",C[i]); }
    printf("\n");

    // free the device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}