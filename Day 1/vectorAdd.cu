#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < numElements){
        C[index] = A[index] + B[index];
    }
}
int main(void){
    // Length of the vector
    const int numElements = 50000;
    // Declare the allocated size for float
    size_t size = numElements * sizeof(float);
    // Allocate host memory for vector A
    float *A = (float *)malloc(size);
    // Allocate the host memory for vector B
    float *B = (float *)malloc(size);
    for(int i=0;i<numElements;i++){
        A[i]=rand() / (float)RAND_MAX;
        B[i] = rand() / (float)RAND_MAX;
        printf("%f %f\n",A[i],B[i]);
    }
    // Allocate the outout vector C
    float *C = (float *)malloc(size);

    // Allocate device input vector
    float *A_d = NULL, *B_d = NULL, *C_d = NULL;
    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    printf("Copy input data from host memory to CUDA device\n");
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>> (A_d, B_d, C_d, numElements);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize(); // Some PTX Error for 4090
    // verify the result
    for(int i=0;i<numElements;++i){
        if(fabs(A[i]+B[i]-C[i])> 1e-6){
            printf("Result verification failed\n");
            // exit(0);
            // printf("%f ", C[i]);
        }else{
            printf("%f ",C[i]);
        }
    }
    printf("\n");
    // Free all the memory in GPU
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;

}